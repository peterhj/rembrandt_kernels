
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdint.h>

__global__ void softmax_kl_loss_fwd_batch_kernel(
    const float *out_act,
    int dim,
    int batch_size,
    const int32_t *label_cats,
    const float *weights,
    const float *targets,
    float *out_loss)
{
  int batch_idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (batch_idx < batch_size) {
    int cat_i = label_cats[batch_idx];
    int idx = cat_i + batch_idx * dim;
    float x = -logf(out_act[idx]) * weights[batch_idx] * targets[batch_idx];
    out_loss[batch_idx] = x;
  }
}

extern "C" void rembrandt_kernel_softmax_kl_loss_fwd_batch(
    const float *out_act,
    int dim,
    int batch_size,
    const int32_t *label_cats,
    const float *weights,
    const float *targets,
    float *out_loss,
    hipStream_t stream)
{
  int n = batch_size;
  softmax_kl_loss_fwd_batch_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(
      out_act, dim, batch_size,
      label_cats,
      weights,
      targets,
      out_loss);
}

__global__ void softmax_kl_loss_bwd_batch_kernel(
    const float *out_act,
    int dim,
    int batch_size,
    const int32_t *label_cats,
    const float *weights,
    const float *targets,
    float *in_delta)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int i = idx % dim;
  int batch_idx = idx / dim;
  if ((i < dim) && (batch_idx < batch_size)) {
    int cat_i = label_cats[batch_idx];
    float dx = out_act[idx];
    if (i == cat_i) {
      dx -= 1.0f;
    }
    dx *= weights[batch_idx] * targets[batch_idx];
    in_delta[idx] = dx;
  }
}

extern "C" void rembrandt_kernel_softmax_kl_loss_bwd_batch(
    const float *out_act,
    int dim,
    int batch_size,
    const int32_t *label_cats,
    const float *weights,
    const float *targets,
    float *in_delta,
    hipStream_t stream)
{
  int n = dim * batch_size;
  softmax_kl_loss_bwd_batch_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(
      out_act, dim, batch_size,
      label_cats,
      weights,
      targets,
      in_delta);
}

__global__ void softmax_r_fwd_batch_kernel(
    const float *in_r_act,
    int dim,
    int batch_size,
    const float *mix_in_r_act,
    float *out_r_act)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int i = idx % dim;
  int batch_idx = idx / dim;
  if ((i < dim) && (batch_idx < batch_size)) {
    /*float x = out_act[idx];
    x *= in_r_act[idx] - mix_in_r_act[batch_idx];*/
    float x = in_r_act[idx] - mix_in_r_act[batch_idx];
    out_r_act[idx] = x;
  }
}

extern "C" void rembrandt_kernel_softmax_r_fwd_batch(
    const float *in_r_act,
    int dim,
    int batch_size,
    const float *mix_in_r_act,
    float *out_r_act,
    hipStream_t stream)
{
  int n = dim * batch_size;
  softmax_r_fwd_batch_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(
      in_r_act,
      dim, batch_size,
      mix_in_r_act,
      out_r_act);
}

__global__ void softmax_kl_loss_r_fwd_batch_kernel(
    const float *out_r_act,
    int dim,
    int batch_size,
    const int32_t *label_cats,
    const float *r_weights,
    float *out_r_loss)
{
  int batch_idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (batch_idx < batch_size) {
    int label_i = label_cats[batch_idx];
    int label_idx = label_i + batch_idx * dim;
    //float x = -out_r_act[label_idx] / out_act[label_idx] * r_weights[batch_idx];
    float x = -out_r_act[label_idx] * r_weights[batch_idx];
    out_r_loss[batch_idx] = x;
  }
}

extern "C" void rembrandt_kernel_softmax_kl_loss_r_fwd_batch(
    const float *out_r_act,
    int dim,
    int batch_size,
    const int32_t *label_cats,
    const float *r_weights,
    float *out_r_loss,
    hipStream_t stream)
{
  int n = batch_size;
  softmax_kl_loss_r_fwd_batch_kernel<<<(n+128-1)/128, 128, 0, stream>>>(
      out_r_act,
      dim, batch_size,
      label_cats,
      r_weights,
      out_r_loss);
}
