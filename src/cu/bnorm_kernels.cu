#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime_api.h>
#include <stdint.h>

__global__ void conv_diag_affine_white_fwd_batch_kernel(
    const float *in_act,
    int spatial_dim,
    int num_channels,
    int batch_size,
    const float *mean,
    const float *istd,
    float *out_act)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int u = idx % spatial_dim;
  int c = (idx / spatial_dim) % num_channels;
  int batch_idx = idx / (spatial_dim * num_channels);
  if (u < spatial_dim && c < num_channels && batch_idx < batch_size) {
    float m = mean[c];
    float is = istd[c];
    float y = is * (in_act[idx] - m);
    out_act[idx] = y;
  }
}

extern "C" void rembrandt_conv_diag_affine_white_fwd_batch(
    const float *in_act,
    int spatial_dim,
    int num_channels,
    int batch_size,
    const float *scale,
    const float *bias,
    float *out_act,
    hipStream_t stream)
{
  int n = spatial_dim * num_channels * batch_size;
  conv_diag_affine_white_fwd_batch_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(
      in_act, spatial_dim, num_channels, batch_size, scale, bias, out_act);
}

__global__ void conv_diag_affine_fwd_batch_kernel(
    const float *in_act,
    int spatial_dim,
    int num_channels,
    int batch_size,
    const float *scale,
    const float *bias,
    float *out_act)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int u = idx % spatial_dim;
  int c = (idx / spatial_dim) % num_channels;
  int batch_idx = idx / (spatial_dim * num_channels);
  if (u < spatial_dim && c < num_channels && batch_idx < batch_size) {
    float gamma = scale[c];
    float beta = bias[c];
    float y = gamma * in_act[idx] + beta;
    out_act[idx] = y;
  }
}

extern "C" void rembrandt_conv_diag_affine_fwd_batch(
    const float *in_act,
    int spatial_dim,
    int num_channels,
    int batch_size,
    const float *scale,
    const float *bias,
    float *out_act,
    hipStream_t stream)
{
  int n = spatial_dim * num_channels * batch_size;
  conv_diag_affine_fwd_batch_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(
      in_act, spatial_dim, num_channels, batch_size, scale, bias, out_act);
}

extern "C" void rembrandt_conv_diag_affine_fwd_inplace_batch(
    float *out_act,
    int spatial_dim,
    int num_channels,
    int batch_size,
    const float *scale,
    const float *bias,
    hipStream_t stream)
{
  int n = spatial_dim * num_channels * batch_size;
  conv_diag_affine_fwd_batch_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(
      out_act, spatial_dim, num_channels, batch_size, scale, bias, out_act);
}

__global__ void conv_diag_affine_bwd_data_batch_kernel(
    const float *in_act,
    const float *out_delta,
    int spatial_dim,
    int num_channels,
    int batch_size,
    const float *scale,
    float *in_delta)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int block_spatial_dim = (spatial_dim+16*32-1)/(16*32);
  int warp_idx = idx % 32;
  int c = (idx / 32) % num_channels;
  int u0 = warp_idx + ((idx / (32 * num_channels)) % block_spatial_dim) * (16*32);
  int batch_idx = idx / (32 * num_channels * block_spatial_dim);
  if (c < num_channels && u0 < spatial_dim && batch_idx < batch_size) {
    float gamma = scale[c];
    int i0 = c * spatial_dim + batch_idx * spatial_dim * num_channels;
    int u_limit = min(spatial_dim, u0 + 16*32);
    for (int u = u0; u < u_limit; u += 32) {
      int i = i0 + u;
      float dy = out_delta[i];
      in_delta[i] = dy * gamma;
      //in_delta[i] += dy * gamma;
      //atomicAdd(&in_delta[i], dy * gamma);
    }
  }
}

extern "C" void rembrandt_conv_diag_affine_bwd_data_batch(
    const float *in_act,
    const float *out_delta,
    int spatial_dim,
    int num_channels,
    int batch_size,
    const float *scale,
    float *in_delta,
    hipStream_t stream)
{
  int block_spatial_dim = (spatial_dim+16*32-1)/(16*32);
  int n = 32 * num_channels * block_spatial_dim * batch_size;
  conv_diag_affine_bwd_data_batch_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(
      in_act, out_delta, spatial_dim, num_channels, batch_size, scale, in_delta);
}

__global__ void conv_diag_affine_bwd_batch_kernel(
    const float *in_act,
    const float *out_delta,
    int spatial_dim,
    int num_channels,
    int batch_size,
    const float *scale,
    float *scale_grad,
    float *bias_grad,
    float *in_delta)
{
  __shared__ float scale_grad_cache[1024+32];
  __shared__ float bias_grad_cache[1024+32];
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int bank_idx = OFFSET_BANK(threadIdx.x);
  int block_spatial_dim = (spatial_dim+16*32-1)/(16*32);
  int warp_idx = idx % 32;
  int c = (idx / 32) % num_channels;
  int u0 = warp_idx + ((idx / (32 * num_channels)) % block_spatial_dim) * (16*32);
  int batch_idx = idx / (32 * num_channels * block_spatial_dim);
  if (c < num_channels && u0 < spatial_dim && batch_idx < batch_size) {
    float gamma = scale[c];
    float d_gamma = 0.0f;
    float d_beta = 0.0f;
    int i0 = c * spatial_dim + batch_idx * spatial_dim * num_channels;
    int u_limit = min(spatial_dim, u0 + 16*32);
    for (int u = u0; u < u_limit; u += 32) {
      int i = i0 + u;
      float dy = out_delta[i];
      d_gamma += dy * in_act[i];
      d_beta += dy;
      in_delta[i] = dy * gamma;
      //in_delta[i] += dy * gamma;
      //atomicAdd(&in_delta[i], dy * gamma);
    }
    scale_grad_cache[bank_idx] = d_gamma;
    bias_grad_cache[bank_idx] = d_beta;
  } else {
    scale_grad_cache[bank_idx] = 0.0f;
    bias_grad_cache[bank_idx] = 0.0f;
  }
  __syncthreads();
  if (c < num_channels && batch_idx < batch_size) {
    if (bank_idx % 2 == 0) {
      scale_grad_cache[bank_idx] += scale_grad_cache[bank_idx+1];
      bias_grad_cache[bank_idx] += bias_grad_cache[bank_idx+1];
    }
    __syncthreads();
    if (bank_idx % 4 == 0) {
      scale_grad_cache[bank_idx] += scale_grad_cache[bank_idx+2];
      bias_grad_cache[bank_idx] += bias_grad_cache[bank_idx+2];
    }
    __syncthreads();
    if (bank_idx % 8 == 0) {
      scale_grad_cache[bank_idx] += scale_grad_cache[bank_idx+4];
      bias_grad_cache[bank_idx] += bias_grad_cache[bank_idx+4];
    }
    __syncthreads();
    if (bank_idx % 16 == 0) {
      scale_grad_cache[bank_idx] += scale_grad_cache[bank_idx+8];
      bias_grad_cache[bank_idx] += bias_grad_cache[bank_idx+8];
    }
    __syncthreads();
    if (bank_idx % 32 == 0 && u0 < spatial_dim) {
      float d_gamma = scale_grad_cache[bank_idx] + scale_grad_cache[bank_idx+16];
      atomicAdd(&scale_grad[c], d_gamma);
      float d_beta = bias_grad_cache[bank_idx] + bias_grad_cache[bank_idx+16];
      atomicAdd(&bias_grad[c], d_beta);
    }
  /*} else {
    __syncthreads();
    __syncthreads();
    __syncthreads();
    __syncthreads();*/
  }
}

extern "C" void rembrandt_conv_diag_affine_bwd_batch(
    const float *in_act,
    const float *out_delta,
    int spatial_dim,
    int num_channels,
    int batch_size,
    const float *scale,
    float *scale_grad,
    float *bias_grad,
    float *in_delta,
    hipStream_t stream)
{
  int block_spatial_dim = (spatial_dim+16*32-1)/(16*32);
  int n = 32 * num_channels * block_spatial_dim * batch_size;
  conv_diag_affine_bwd_batch_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(
      in_act, out_delta, spatial_dim, num_channels, batch_size, scale, scale_grad, bias_grad, in_delta);
}

__global__ void conv_diag_linear_bwd_batch_kernel(
    const float *in_act,
    const float *out_delta,
    int spatial_dim,
    int num_channels,
    int batch_size,
    const float *scale,
    float *scale_grad,
    float *in_delta)
{
  __shared__ float scale_grad_cache[1024+32];
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int bank_idx = OFFSET_BANK(threadIdx.x);
  int block_spatial_dim = (spatial_dim+4*32-1)/(4*32);
  int c = (idx / 32) % num_channels;
  int u0 = ((idx / (32 * num_channels)) % block_spatial_dim) * (4*32);
  int batch_idx = idx / (32 * num_channels * block_spatial_dim);
  if (c < num_channels && u0 < spatial_dim && batch_idx < batch_size) {
    float gamma = scale[c];
    float d_gamma = 0.0f;
    int i0 = c * spatial_dim + batch_idx * spatial_dim * num_channels;
    int u_limit = min(spatial_dim, u0 + 4*32);
    for (int u = u0; u < u_limit; u += 32) {
      int i = i0 + u;
      float dy = out_delta[i];
      d_gamma += dy * in_act[i];
      in_delta[i] += dy * gamma;
    }
    scale_grad_cache[bank_idx] = d_gamma;
  } else {
    scale_grad_cache[bank_idx] = 0.0f;
  }
  __syncthreads();
  if (c < num_channels && batch_idx < batch_size) {
    if (bank_idx % 2 == 0) {
      scale_grad_cache[bank_idx] += scale_grad_cache[bank_idx+1];
    }
    __syncthreads();
    if (bank_idx % 4 == 0) {
      scale_grad_cache[bank_idx] += scale_grad_cache[bank_idx+2];
    }
    __syncthreads();
    if (bank_idx % 8 == 0) {
      scale_grad_cache[bank_idx] += scale_grad_cache[bank_idx+4];
    }
    __syncthreads();
    if (bank_idx % 16 == 0) {
      scale_grad_cache[bank_idx] += scale_grad_cache[bank_idx+8];
    }
    __syncthreads();
    if (bank_idx % 32 == 0 && u0 < spatial_dim) {
      float d_gamma = scale_grad_cache[bank_idx] + scale_grad_cache[bank_idx+16];
      atomicAdd(&scale_grad[c], d_gamma);
    }
  }
}

__global__ void conv_bnorm_bwd_var_batch_kernel(
    const float *in_act,
    int spatial_dim,
    int num_channels,
    int batch_size,
    const float *out_delta,
    const float *mean,
    const float *var,
    float epsilon,
    float *var_grad)
{
  __shared__ float d_sigma_cache[1024+32];
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int bank_idx = OFFSET_BANK(threadIdx.x);
  int block_spatial_dim = (spatial_dim+16*32-1)/(16*32);
  int warp_idx = idx % 32;
  int c = (idx / 32) % num_channels;
  int u0 = warp_idx + ((idx / (32 * num_channels)) % block_spatial_dim) * (16*32);
  int batch_idx = idx / (32 * num_channels * block_spatial_dim);
  if (c < num_channels && u0 < spatial_dim && batch_idx < batch_size) {
    float mu = mean[c];
    float sigma = var[c];
    float d_sigma = 0.0f;
    int i0 = c * spatial_dim + batch_idx * spatial_dim * num_channels;
    int u_limit = min(spatial_dim, u0 + 16*32);
    for (int u = u0; u < u_limit; u += 32) {
      int i = i0 + u;
      d_sigma += out_delta[i] * -0.5f * rsqrtf(sigma + epsilon) / (sigma + epsilon) * (in_act[i] - mu);
    }
    d_sigma_cache[bank_idx] = d_sigma;
  } else {
    d_sigma_cache[bank_idx] = 0.0f;
  }
  __syncthreads();
  if (c < num_channels && batch_idx < batch_size) {
    if (bank_idx % 2 == 0) {
      d_sigma_cache[bank_idx] += d_sigma_cache[bank_idx+1];
    }
    __syncthreads();
    if (bank_idx % 4 == 0) {
      d_sigma_cache[bank_idx] += d_sigma_cache[bank_idx+2];
    }
    __syncthreads();
    if (bank_idx % 8 == 0) {
      d_sigma_cache[bank_idx] += d_sigma_cache[bank_idx+4];
    }
    __syncthreads();
    if (bank_idx % 16 == 0) {
      d_sigma_cache[bank_idx] += d_sigma_cache[bank_idx+8];
    }
    __syncthreads();
    if (bank_idx % 32 == 0 && u0 < spatial_dim) {
      float d_sigma = d_sigma_cache[bank_idx] + d_sigma_cache[bank_idx+16];
      atomicAdd(&var_grad[c], d_sigma);
    }
  }
}

__global__ void conv_bnorm_bwd_mean_batch_kernel(
    const float *in_act,
    int spatial_dim,
    int num_channels,
    int batch_size,
    const float *out_delta,
    const float *mean,
    const float *var,
    const float *var_grad,
    float epsilon,
    float *mean_grad)
{
}

__global__ void conv_bnorm_bwd_data_batch_kernel(
    const float *in_act,
    int spatial_dim,
    int num_channels,
    int batch_size,
    const float *out_delta,
    const float *mean,
    const float *mean_grad,
    const float *var,
    const float *var_grad,
    float epsilon,
    float *in_delta)
{
}

extern "C" void rembrandt_conv_bnorm_bwd_batch(
    const float *in_act,
    int spatial_dim,
    int num_channels,
    int batch_size,
    const float *out_delta,
    const float *mean,
    const float *var,
    float epsilon,
    float *mean_grad,
    float *var_grad,
    float *in_delta,
    hipStream_t stream)
{
}
