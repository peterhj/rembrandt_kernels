
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdint.h>

__device__ float bicubic_w0(float a) {
  return (1.0f/6.0f)*(a*(a*(-a + 3.0f) - 3.0f) + 1.0f);
}

__device__ float bicubic_w1(float a) {
  return (1.0f/6.0f)*(a*a*(3.0f*a - 6.0f) + 4.0f);
}

__device__ float bicubic_w2(float a) {
  return (1.0f/6.0f)*(a*(a*(-3.0f*a + 3.0f) + 3.0f) + 1.0f);
}

__device__ float bicubic_w3(float a) {
  return (1.0f/6.0f)*(a*a*a);
}

__device__ float catrom_w0(float a) {
    //return -0.5f*a + a*a - 0.5f*a*a*a;
    return a*(-0.5f + a*(1.0f - 0.5f*a));
}

__device__ float catrom_w1(float a) {
    //return 1.0f - 2.5f*a*a + 1.5f*a*a*a;
    return 1.0f + a*a*(-2.5f + 1.5f*a);
}

__device__ float catrom_w2(float a) {
    //return 0.5f*a + 2.0f*a*a - 1.5f*a*a*a;
    return a*(0.5f + a*(2.0f - 1.5f*a));
}

__device__ float catrom_w3(float a) {
    //return -0.5f*a*a + 0.5f*a*a*a;
    return a*a*(-0.5f + 0.5f*a);
}

/*__device__ float mitchell_w0(float a) {
  float b = absf(a);
  return
      (b < 1.0f) * () +
      (b >= 1.0f) * (b < 2.0f) * ();
}*/

__device__ float image3_tex2d_clamp(const float *pixels, int width, int height, int u, int v, int c) {
  int clamp_u = min(max(0, u), width-1);
  int clamp_v = min(max(0, v), height-1);
  return pixels[clamp_u + clamp_v * width + c * width * height];
}

__device__ float image3_bicubic_filter(
    float x,
    float a0,
    float a1,
    float a2,
    float a3)
{
  float r = a0 * bicubic_w0(x);
  r += a1 * bicubic_w1(x);
  r += a2 * bicubic_w2(x);
  r += a3 * bicubic_w3(x);
  return r;
}

__device__ float image3_bicubic_interpolate(
    const float *pixels,
    int width,
    int height,
    float u,
    float v,
    int c)
{
  u -= 0.5f;
  v -= 0.5f;
  float px = floorf(u);
  float py = floorf(v);
  float fx = u - px;
  float fy = v - py;
  int ipx = (int)px;
  int ipy = (int)py;
  return image3_bicubic_filter(fy,
      image3_bicubic_filter(fx,
          image3_tex2d_clamp(pixels, width, height, ipx-1, ipy-1, c),
          image3_tex2d_clamp(pixels, width, height, ipx,   ipy-1, c),
          image3_tex2d_clamp(pixels, width, height, ipx+1, ipy-1, c),
          image3_tex2d_clamp(pixels, width, height, ipx+2, ipy-1, c)),
      image3_bicubic_filter(fx,
          image3_tex2d_clamp(pixels, width, height, ipx-1, ipy,   c),
          image3_tex2d_clamp(pixels, width, height, ipx,   ipy,   c),
          image3_tex2d_clamp(pixels, width, height, ipx+1, ipy,   c),
          image3_tex2d_clamp(pixels, width, height, ipx+2, ipy,   c)),
      image3_bicubic_filter(fx,
          image3_tex2d_clamp(pixels, width, height, ipx-1, ipy+1, c),
          image3_tex2d_clamp(pixels, width, height, ipx,   ipy+1, c),
          image3_tex2d_clamp(pixels, width, height, ipx+1, ipy+1, c),
          image3_tex2d_clamp(pixels, width, height, ipx+2, ipy+1, c)),
      image3_bicubic_filter(fx,
          image3_tex2d_clamp(pixels, width, height, ipx-1, ipy+2, c),
          image3_tex2d_clamp(pixels, width, height, ipx,   ipy+2, c),
          image3_tex2d_clamp(pixels, width, height, ipx+1, ipy+2, c),
          image3_tex2d_clamp(pixels, width, height, ipx+2, ipy+2, c)));
}

__global__ void image3_bicubic_scale_kernel(
    const float *in_pixels,
    int in_width,
    int in_height,
    int channels,
    float *out_pixels,
    int out_width,
    int out_height)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int x = idx % out_width;
  int y = (idx / out_width) % out_height;
  int c = idx / (out_width * out_height);

  if ((x < out_width) && (y < out_height) && (c < channels)) {
    float u = ((float)x) / ((float)out_width) * ((float)in_width);
    float v = ((float)y) / ((float)out_height) * ((float)in_height);

    float interp_value = image3_bicubic_interpolate(in_pixels, in_width, in_height, u, v, c);

    out_pixels[x + y * out_width + c * out_width * out_height] = interp_value;
  }
}

extern "C" void rembrandt_kernel_image3_bicubic_scale(
    const float *in_pixels,
    int in_width,
    int in_height,
    int channels,
    float *out_pixels,
    int out_width,
    int out_height,
    hipStream_t stream)
{
  int n = out_width * out_height * channels;
  image3_bicubic_scale_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(
      in_pixels,
      in_width,
      in_height,
      channels,
      out_pixels,
      out_width,
      out_height);
}

__device__ float image3_catmullrom_filter(
    float x,
    float a0,
    float a1,
    float a2,
    float a3)
{
  float r = a0 * catrom_w0(x);
  r += a1 * catrom_w1(x);
  r += a2 * catrom_w2(x);
  r += a3 * catrom_w3(x);
  return r;
}

__device__ float image3_catmullrom_interpolate(
    const float *pixels,
    int width,
    int height,
    float u,
    float v,
    int c)
{
  u -= 0.5f;
  v -= 0.5f;
  float px = floorf(u);
  float py = floorf(v);
  float fx = u - px;
  float fy = v - py;
  int ipx = (int)px;
  int ipy = (int)py;
  return image3_catmullrom_filter(fy,
      image3_catmullrom_filter(fx,
          image3_tex2d_clamp(pixels, width, height, ipx-1, ipy-1, c),
          image3_tex2d_clamp(pixels, width, height, ipx,   ipy-1, c),
          image3_tex2d_clamp(pixels, width, height, ipx+1, ipy-1, c),
          image3_tex2d_clamp(pixels, width, height, ipx+2, ipy-1, c)),
      image3_catmullrom_filter(fx,
          image3_tex2d_clamp(pixels, width, height, ipx-1, ipy,   c),
          image3_tex2d_clamp(pixels, width, height, ipx,   ipy,   c),
          image3_tex2d_clamp(pixels, width, height, ipx+1, ipy,   c),
          image3_tex2d_clamp(pixels, width, height, ipx+2, ipy,   c)),
      image3_catmullrom_filter(fx,
          image3_tex2d_clamp(pixels, width, height, ipx-1, ipy+1, c),
          image3_tex2d_clamp(pixels, width, height, ipx,   ipy+1, c),
          image3_tex2d_clamp(pixels, width, height, ipx+1, ipy+1, c),
          image3_tex2d_clamp(pixels, width, height, ipx+2, ipy+1, c)),
      image3_catmullrom_filter(fx,
          image3_tex2d_clamp(pixels, width, height, ipx-1, ipy+2, c),
          image3_tex2d_clamp(pixels, width, height, ipx,   ipy+2, c),
          image3_tex2d_clamp(pixels, width, height, ipx+1, ipy+2, c),
          image3_tex2d_clamp(pixels, width, height, ipx+2, ipy+2, c)));
}

__global__ void image3_catmullrom_scale_kernel(
    const float *in_pixels,
    int in_width,
    int in_height,
    int channels,
    float *out_pixels,
    int out_width,
    int out_height)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int x = idx % out_width;
  int y = (idx / out_width) % out_height;
  int c = idx / (out_width * out_height);

  if ((x < out_width) && (y < out_height) && (c < channels)) {
    float u = ((float)x) / ((float)out_width) * ((float)in_width);
    float v = ((float)y) / ((float)out_height) * ((float)in_height);

    float interp_value = image3_catmullrom_interpolate(in_pixels, in_width, in_height, u, v, c);

    out_pixels[x + y * out_width + c * out_width * out_height] = interp_value;
  }
}

extern "C" void rembrandt_kernel_image3_catmullrom_scale(
    const float *in_pixels,
    int in_width,
    int in_height,
    int channels,
    float *out_pixels,
    int out_width,
    int out_height,
    hipStream_t stream)
{
  int n = out_width * out_height * channels;
  image3_catmullrom_scale_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(
      in_pixels,
      in_width,
      in_height,
      channels,
      out_pixels,
      out_width,
      out_height);
}

__device__ float image3_2x2_bilinear_interpolate(
    const float *pixels,
    int width,
    int height,
    float u,
    float v,
    int c)
{
  u -= 0.5f;
  v -= 0.5f;
  float px = floorf(u);
  float py = floorf(v);
  int ipx = (int)px;
  int ipy = (int)py;
  return 0.25 * (
      image3_tex2d_clamp(pixels, width, height, ipx,   ipy,   c) +
      image3_tex2d_clamp(pixels, width, height, ipx+1, ipy,   c) +
      image3_tex2d_clamp(pixels, width, height, ipx,   ipy+1, c) +
      image3_tex2d_clamp(pixels, width, height, ipx+1, ipy+1, c));
}

__global__ void image3_2x2_bilinear_scale_kernel(
    const float *in_pixels,
    int in_width,
    int in_height,
    int channels,
    float *out_pixels,
    int out_width,
    int out_height)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int x = idx % out_width;
  int y = (idx / out_width) % out_height;
  int c = idx / (out_width * out_height);

  if ((x < out_width) && (y < out_height) && (c < channels)) {
    float u = ((float)x) / ((float)out_width) * ((float)in_width);
    float v = ((float)y) / ((float)out_height) * ((float)in_height);

    float interp_value = image3_2x2_bilinear_interpolate(in_pixels, in_width, in_height, u, v, c);

    out_pixels[x + y * out_width + c * out_width * out_height] = interp_value;
  }
}

extern "C" void rembrandt_kernel_image3_2x2_bilinear_scale(
    const float *in_pixels,
    int in_width,
    int in_height,
    int channels,
    float *out_pixels,
    int out_width,
    int out_height,
    hipStream_t stream)
{
  int n = out_width * out_height * channels;
  image3_2x2_bilinear_scale_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(
      in_pixels,
      in_width,
      in_height,
      channels,
      out_pixels,
      out_width,
      out_height);
}
