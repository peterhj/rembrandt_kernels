#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime_api.h>

__global__ void map_exp_kernel(
    float *x,
    int n)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < n) {
    x[i] = expf(x[i]);
  }
}

extern "C" void rembrandt_kernel_map_exp(
    float *x,
    int n,
    hipStream_t stream)
{
  map_exp_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(x, n);
  CUDA_POST_KERNEL_CHECK;
}

__global__ void map_subtract_scalar_kernel(
    float *x,
    int n,
    float *scalar)
{
  __shared__ float c[1];
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (threadIdx.x == 0) {
    c[0] = *scalar;
  }
  __syncthreads();
  if (i < n) {
    x[i] -= c[0];
  }
}

extern "C" void rembrandt_kernel_map_subtract_scalar(
    float *x,
    int n,
    float *scalar,
    hipStream_t stream)
{
  map_subtract_scalar_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(x, n, scalar);
  CUDA_POST_KERNEL_CHECK;
}

__global__ void map_divide_scalar_kernel(
    float *x,
    int n,
    float *scalar)
{
  __shared__ float c[1];
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (threadIdx.x == 0) {
    c[0] = *scalar;
  }
  __syncthreads();
  if (i < n) {
    x[i] /= c[0];
  }
}

extern "C" void rembrandt_kernel_map_divide_scalar(
    float *x,
    int n,
    float *scalar,
    hipStream_t stream)
{
  map_divide_scalar_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(x, n, scalar);
  CUDA_POST_KERNEL_CHECK;
}

__global__ void map_relu_activation_kernel(
    int n,
    float *x)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < n) {
    float x_i = x[i];
    x[i] = fmaxf(0.0, x_i);
  }
}

extern "C" void rembrandt_kernel_map_relu_activation(
    int n,
    float *x,
    hipStream_t stream)
{
  /*dim3 block_dim(CUDA_BLOCK_DIM_1D(n));
  dim3 grid_dim(CUDA_GRID_DIM_1D(n));
  map_relu_activation_kernel<<<grid_dim, block_dim, 0, stream>>>(n, x);*/
  map_relu_activation_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(n, x);
  CUDA_POST_KERNEL_CHECK;
}

__global__ void map_relu_activation_backprop_kernel(
    const float *z,
    int n,
    float *delta)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < n) {
    delta[i] = delta[i] * (z[i] > 0.0);
  }
}

extern "C" void rembrandt_kernel_map_relu_activation_backprop(
    const float *z,
    int n,
    float *delta,
    hipStream_t stream)
{
  map_relu_activation_backprop_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(z, n, delta);
  CUDA_POST_KERNEL_CHECK;
}

__global__ void map_sigmoid_activation_kernel(
    int n,
    float *x)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < n) {
    float x_i = x[i];
    x[i] = 1.0 / (1.0 + expf(-x_i));
  }
}

extern "C" void rembrandt_kernel_map_sigmoid_activation(
    int n,
    float *x,
    hipStream_t stream)
{
  /*dim3 block_dim(CUDA_BLOCK_DIM_1D(n));
  dim3 grid_dim(CUDA_GRID_DIM_1D(n));
  map_sigmoid_activation_kernel<<<grid_dim, block_dim, 0, stream>>>(n, x);*/
  map_sigmoid_activation_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(n, x);
  CUDA_POST_KERNEL_CHECK;
}

extern "C" void rembrandt_kernel_map_kahan_sum_update(
    int n,
    const float *x,
    float *y_sum,
    float *y_err,
    hipStream_t stream)
{
  assert(0 && "unimplemented!");
}

extern "C" void rembrandt_kernel_map_kahan_sum_finish(
    int n,
    const float *y_sum,
    const float *y_err,
    float *s,
    hipStream_t stream)
{
  assert(0 && "unimplemented!");
}

/*__global__ void map_softmax_cross_entropy_kernel(
    int n
    )
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < n) {
  }
}

extern "C" void rembrandt_kernel_map_softmax_cross_entropy(
    hipStream_t stream)
{
  map_softmax_cross_entropy_kernel<<<, , 0, stream>>>();
  CUDA_POST_KERNEL_CHECK;
}*/

__global__ void map_softmax_cross_entropy_backprop_kernel(
    const float *z,
    int n,
    int truth_label,
    float *delta)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < n) {
    if (i == truth_label) {
      delta[i] = z[i] - 1.0;
    } else {
      delta[i] = z[i];
    }
  }
}

extern "C" void rembrandt_kernel_map_softmax_cross_entropy_backprop(
    const float *z,
    int n,
    int truth_label,
    float *delta,
    hipStream_t stream)
{
  map_softmax_cross_entropy_backprop_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(z, n, truth_label, delta);
  CUDA_POST_KERNEL_CHECK;
}

__global__ void map_dropout_kernel(
    const float *x,
    int n,
    float threshold,
    float scale,
    const float *rand,
    float *z,
    int *mask)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < n) {
    int m = rand[i] > threshold;
    z[i] = scale * x[i] * m;
    mask[i] = m;
  }
}

extern "C" void rembrandt_kernel_map_dropout(
    const float *x,
    int n,
    float threshold,
    float scale,
    const float *rand,
    float *z,
    int *mask,
    hipStream_t stream)
{
  map_dropout_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(x, n, threshold, scale, rand, z, mask);
  CUDA_POST_KERNEL_CHECK;
}

__global__ void map_dropout_backprop_kernel(
    const float *z,
    int n,
    float threshold,
    float scale,
    const int *mask,
    float *delta)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < n) {
    delta[i] = scale * z[i] * mask[i];
  }
}

extern "C" void rembrandt_kernel_map_dropout_backprop(
    const float *z,
    int n,
    float threshold,
    float scale,
    const int *mask,
    float *delta,
    hipStream_t stream)
{
  map_dropout_backprop_kernel<<<(n+1024-1)/1024, 1024, 0, stream>>>(z, n, threshold, scale, mask, delta);
  CUDA_POST_KERNEL_CHECK;
}
